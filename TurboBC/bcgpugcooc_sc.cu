#include "hip/hip_runtime.h"
/*
 * This source code is distributed under the terms defined  
 * in the file bcugcsccooc_main.c of this source distribution.
 */
/* 
 *  Betweenness centrality (BC)  
 *  Single precision (float data type) 
 *  TurboBC:bcgpugcooc_sc.cu
 * 
 *  This program computes the GPU-based parallel BC
 *  (scalar) for unweighted graphs represented 
 *  by sparse adjacency matrices in the COOC format.
 *
 */

#include <cstdlib>
#include <iostream>
#include <cassert>
#include <cmath>

//includes CUDA project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "bcgpug.cuh"
extern "C"{
                 #include "bcug.h"
}


/*************************prototype kernels*************************************/
__global__ void spMVBFSCoocScKernel (int *I_d,int *J_d,int *ft_d,int *f_d,
				     float *sigma_d,int d,int r,int nz);
/******************************************************************************/
__global__ void spMVBCoocScKernel (int *I_d,int *J_d,float *delta_ut_d,
				   float *delta_u_d,int nz);
/******************************************************************************/
__global__ void  bfsFunctCoocKernel (int *f_d,int *ft_d,float *sigma_d,int *S_d,
	   	 		     int *c,int n,int d);
/******************************************************************************/

/* 
 * function to compute a gpu-based parallel BC (scalar) for unweighted graphs 
 * represented by sparse adjacency matrices in COOC format. 
 *  
 */
int  bc_gpu_ug_cooc_sc (int *I_h,int *J_h,int *S_h,float *sigma_h,float *bc_h,
			int nr,int rs,int nz,int n,int repetition){
  float t_H_to_D_I;
  float t_H_to_D_J;
  float t_D_to_H_sigma;
  float t_D_to_H_S;
  float t_D_to_H_bc;
  float t_bfs_spmv = 0.0;
  float t_bfs_spmv_t= 0.0;
  float t_bfsfunctions = 0.0;
  float t_bfsfunctions_t = 0.0;
  float t_bfs_sum = 0.0;
  float t_bfs_avg = 0.0;
  float t_allocate = 0.0;
  float t_allocate_t = 0.0;
  float t_delta_u = 0.0;
  float t_delta_u_t = 0.0;
  float t_bc_spmv = 0.0;
  float t_bc_spmv_t = 0.0;
  float t_delta = 0.0;
  float t_delta_t = 0.0;
  float t_delta_sum = 0.0;
  float t_delta_avg = 0.0;
  float t_bc = 0.0;
  float total_BC_t = 0.0;
  int i,r,d,dimGrid,dimGridspmv;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float *delta_d;
  float *delta_u_d;
  float *delta_ut_d;
  int *ft_d;
  int *f_d;

  /*Allocate device memory for the vector J_d */
  int *J_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&J_d),sizeof(*J_d)*nz));
  hipEventRecord(stop);

  /*Copy host memory (J_h) to device memory (J_d)*/
  hipEventRecord(start);
  checkCudaErrors(hipMemcpy(J_d,J_h,nz*sizeof(*J_d),hipMemcpyHostToDevice));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&t_H_to_D_J,start,stop);

  /*Allocate device memory for the vector I_d */
  int *I_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&I_d),sizeof(*I_d)*nz));
  /*Copy host memory (I_h) to device memory (I_d)*/
  hipEventRecord(start);
  checkCudaErrors(hipMemcpy(I_d,I_h,nz*sizeof(*I_d),hipMemcpyHostToDevice));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&t_H_to_D_I,start,stop);

  /*Allocate device memory for the vector S_d. */
  int *S_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&S_d),sizeof(*S_d)*n));

  /*Allocate device memory for the vector sigma_d */
  float *sigma_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&sigma_d),sizeof(*sigma_d)*n));
  int *c;
  checkCudaErrors(hipMallocManaged(reinterpret_cast<void **>(&c),sizeof(*c)));

  /*Allocate device memory for the vector bc_d and set bc_d to zero*/
  float *bc_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&bc_d),sizeof(*bc_d)*n));
  checkCudaErrors(hipMemset(bc_d,0.0,sizeof(*bc_d)*n));

  dimGrid = (n + THREADS_PER_BLOCK)/THREADS_PER_BLOCK;
  dimGridspmv = (nz + THREADS_PER_BLOCK)/THREADS_PER_BLOCK;
  /*computing BC */
  for (r=0; r<nr; r++){

    if (nr == 1) r = rs;
    /*Allocate device memory for the vector delta_d*/
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&delta_d),sizeof(*delta_d)*n));

    /*computing BFS */
    for (i = 0; i<repetition; i++){

      /*Allocate device memory for the vector ft_d*/
      checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&ft_d),sizeof(*ft_d)*n));
      /*Allocate device memory for the vector f_d and set f_d to zero*/
      checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&f_d),sizeof(*f_d)*n));
      checkCudaErrors(hipMemset(f_d,0,sizeof(*f_d)*n));
      checkCudaErrors(hipMemset(sigma_d,0.0,sizeof(*sigma_d)*n));
      checkCudaErrors(hipMemset(S_d,0,sizeof(*S_d)*n));

      *c = 1;
      d = 0;
      while (*c){
	d = d + 1;
	*c = 0;
	hipEventRecord(start);
	checkCudaErrors(hipMemset(ft_d,0,sizeof(*ft_d)*n));
	spMVBFSCoocScKernel <<<dimGridspmv,THREADS_PER_BLOCK>>> (I_d,J_d,ft_d,f_d,sigma_d,d,r,nz);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t_bfs_spmv,start,stop);
	t_bfs_spmv_t += t_bfs_spmv;

	hipEventRecord(start);
	bfsFunctCoocKernel <<<dimGrid,THREADS_PER_BLOCK>>> (f_d,ft_d,sigma_d,S_d,c,n,d);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t_bfsfunctions,start,stop);
	t_bfsfunctions_t += t_bfsfunctions;
	
	t_bfs_sum += t_bfs_spmv + t_bfsfunctions;
      }

      /*freeing and allocating memory for BC while loop  */
      hipEventRecord(start);
      //free memory of f_d and ft_d vectors
      checkCudaErrors(hipFree(f_d));
      checkCudaErrors(hipFree(ft_d));
      /*set delta_d to zero*/
      checkCudaErrors(hipMemset(delta_d,0.0,sizeof(*delta_d)*n));
      /*Allocate device memory for the vector delta_u_d*/
      checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&delta_u_d),sizeof(*delta_u_d)*n));
      /*Allocate device memory for the vector delta_ut_d*/
      checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&delta_ut_d),sizeof(*delta_ut_d)*n));
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t_allocate,start,stop);
      t_allocate_t += t_allocate;

      /*computing delta with while loop  */
      d = d-1;
      while (d > 1){

	hipEventRecord(start);
	deltaUKernel <<<dimGrid,THREADS_PER_BLOCK>>> (S_d,delta_d,delta_u_d,sigma_d,n,d);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t_delta_u,start,stop);
	t_delta_u_t += t_delta_u;

	hipEventRecord(start);
	checkCudaErrors(hipMemset(delta_ut_d,0.0,sizeof(*delta_ut_d)*n));
	spMVBCoocScKernel <<<dimGridspmv,THREADS_PER_BLOCK>>> (I_d,J_d,delta_ut_d,delta_u_d,nz);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t_bc_spmv,start,stop);
	t_bc_spmv_t += t_bc_spmv;

	hipEventRecord(start);
	deltaKernel <<<dimGrid,THREADS_PER_BLOCK>>> (S_d,delta_d,delta_ut_d,sigma_d,n,d);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t_delta,start,stop);
	t_delta_t += t_delta;

	d = d-1;
	t_delta_sum += t_delta_u + t_bc_spmv + t_delta;
      }
      checkCudaErrors(hipFree(delta_u_d));
      checkCudaErrors(hipFree(delta_ut_d));
    }//end repetition
    hipEventRecord(start);
    bcKernel <<<dimGrid,THREADS_PER_BLOCK>>> (bc_d,delta_d,r,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t_bc,start,stop);

    checkCudaErrors(hipFree(delta_d));
  }//end for loop
  t_bfs_avg = t_bfs_sum/repetition;
  t_delta_avg = t_delta_sum/repetition;
  total_BC_t =  t_bfs_avg + t_delta_avg + t_allocate/repetition + t_bc;

  /*Copy device memory (sigma_d) to host memory (sigma_h)*/
  hipEventRecord(start);
  checkCudaErrors(hipMemcpy(sigma_h,sigma_d, n*sizeof(*sigma_d),hipMemcpyDeviceToHost));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&t_D_to_H_sigma,start,stop);

  /*Copy device memory (S_d) to host memory (S_h)*/
  hipEventRecord(start);
  checkCudaErrors(hipMemcpy(S_h,S_d, n*sizeof(*S_d),hipMemcpyDeviceToHost));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&t_D_to_H_S,start,stop);

  /*Copy device memory (bc_d) to host memory (bc_h)*/
  hipEventRecord(start);
  checkCudaErrors(hipMemcpy(bc_h,bc_d, n*sizeof(*bc_d),hipMemcpyDeviceToHost));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&t_D_to_H_bc,start,stop);

  int print_t = 1;
  if (print_t){
    printf("\nbcgpugcooc_sc::bc_gpu_ug_cooc_sc:time J_h to J_d = %lfms \n",t_H_to_D_J);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time I_h to I_d = %lfms \n",t_H_to_D_I);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time f <-- A'f  = %lfms \n",t_bfs_spmv_t/repetition);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time time bfs functions  = %lfms \n", t_bfsfunctions_t/repetition);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:average time BFS  = %lfms \n",t_bfs_avg);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:average time BFS/vertex  = %lfms \n",t_bfs_avg/nr);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time to allocate memory for BC stage = %lfms \n",t_allocate/repetition);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time delta_u <-- (1+delta)/sigma  = %lfms \n",t_delta_u_t/repetition);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time delta_ut <-- A'delta_u  = %lfms \n",t_bc_spmv_t/repetition);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time delta <-- delta + delta_ut*sigma  = %lfms \n",t_delta_t/repetition);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:average time delta   = %lfms \n",t_delta_avg);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:average time delta/vertex   = %lfms \n",t_delta_avg/nr);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time bc <-- bc +delta/2 = %lfms \n",t_bc);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:total BC time = %lfms \n",total_BC_t);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:total BC time/vertex = %lfms \n",total_BC_t/nr);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time sigma_d to sigma_h = %lfms \n",t_D_to_H_sigma);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time S_d to S_h = %lfms \n",t_D_to_H_S);
    printf("bcgpugcooc_sc::bc_gpu_ug_cooc_sc:time bc_d to bc_h = %lfms \n",t_D_to_H_bc);
  }

  /*cleanup memory*/
  checkCudaErrors(hipFree(I_d));
  checkCudaErrors(hipFree(J_d));
  checkCudaErrors(hipFree(sigma_d));
  checkCudaErrors(hipFree(S_d));
  checkCudaErrors(hipFree(bc_d));
  checkCudaErrors(hipFree(c));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  return 0;
}//end bfs_gpu_ug_cooc_sc


////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
/* 
 * if d = 1, initialize f(r) and sigma(r),
 * compute the gpu-based parallel sparse matrix-vector multiplication    
 * for sparse matrices in the CSC format, representing unweighted 
 * graphs. 
 */
__global__
void spMVBFSCoocScKernel (int *I_d,int *J_d,int *ft_d,int *f_d,
			  float *sigma_d,int d,int r,int nz){

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < nz){
    //if d = 1, initialize f(r) and sigma(r)
    if (d == 1){
      f_d[r] = 1;
      sigma_d[r] = 1.0;
    }
    //compute spmv
    if (f_d[I_d[i]] != 0) {
      int f = f_d[I_d[i]];
      atomicAdd(&ft_d[J_d[i]],f);
    }
  }
}//end spMVBFSCoocscKernel

/******************************************************************************/
/*
 * computes the delta_ut vector with a SpMV multiplication operation.
 */
__global__
void spMVBCoocScKernel (int *I_d,int *J_d,float *delta_ut_d,
			float *delta_u_d,int nz){

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < nz){
    //compute spmv
    if (delta_u_d[I_d[i]] != 0) {
      float delta = delta_u_d[I_d[i]];
      atomicAdd(&delta_ut_d[J_d[i]],delta);
    }
  }
}//end spMVBCoocscKernel

/******************************************************************************/
/*
 * if sigma(i) == 0, assign vector ft_d to vector f_d,
 * check that the vector f_d  has at least one nonzero element
 * add the vector f to vector sigma.
 */
__global__
void bfsFunctCoocKernel (int *f_d,int *ft_d,float *sigma_d,int *S_d,
     			 int *c,int n,int d){

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n){
    f_d[i] = 0;
    if (sigma_d[i] < 0.01) f_d[i] = ft_d[i];
    if (f_d[i] > 0.9){
      *c = 1;
      sigma_d[i] += f_d[i];
      S_d[i] = d;
    }
  }

}//end  bfsFunctCoocKernel
